#include "CudaKernel.hpp"
#include "EventTimer.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <iostream>
#include <map>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>       /* time */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


thrust::host_vector<float> preload_data_host;
thrust::device_vector<float> preload_data_device;

// convert a linear index to a row index
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T>
{
  T C; // number of columns
  
  __host__ __device__
  linear_index_to_row_index(T C) : C(C) {}

  __host__ __device__
  T operator()(T i)
  {
    return i / C;
  }
};
struct is_valid_column : thrust::unary_function<bool, bool>
{
    __host__ __device__
    bool operator()(const bool &x)
    {
        return x;
    }
};

void cuda_preload_data(float *data){
	preload_data_device.clear();
}

void color_pack(const std::vector<int>& indices,const unsigned int& R, const unsigned int& C,float *colors,const float *data){
	unsigned int NEWC = C;
	unsigned int RC = R*C;
	
	//printf("Color pack.  R: %d  C: %d\n",R,C);
	//printf("Inactive Subject Count: %d\n",indices.size());
	// generate an index map for faster lookup
	std::map<int,char> index_map;
	{
		for(auto index = indices.begin(); index != indices.end(); ++index){
			index_map[(*index)] = 0;
		}
		NEWC -= index_map.size();	//remove K columns from total
	}
	//printf("Index map created.  Size: %d  New column size: %d\n",index_map.size(),NEWC);


	thrust::device_vector<float> array(R * NEWC);
	thrust::host_vector<float> array_host(R * NEWC);
	//printf("created array: %d\n",array.size());
	//printf("remove %d columns\n",index_map.size());

	//Common::EventTimer ev("filter");
	//ev.BeforeEvent();
	{	// faster filter?

		// quick filter uses column oriented data struction were data are stored .  
		int cidx = 0,nc = 0;
		while(cidx < RC){
			int c = cidx/R;
			if(index_map.find(c) != index_map.end()){
				cidx += R;
				continue;
			}
			

			int ridx = nc;
			for(int i = 0; i < R; i++){
				array_host[ridx] = data[cidx + i];
				ridx += NEWC;
			}
			cidx += R;
			nc += 1;
		}
		
	}
	//ev.AfterEvent();
	//ev.Display();
	array = array_host;

	//Common::EventTimer ev2("reduce");
	//ev2.BeforeEvent();

	thrust::device_vector<float> row_sums(R);
	thrust::device_vector<float> row_indices(R);
	{	//reduce
		thrust::reduce_by_key(
			thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(NEWC)),
			thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(NEWC)) + (R*NEWC),
			array.begin(),
			row_indices.begin(),
			row_sums.begin(),
			thrust::equal_to<float>(),
			thrust::plus<float>()
		);
	}
	//ev2.AfterEvent();
	//ev2.Display();

	//Common::EventTimer ev3("repack");
	//ev3.BeforeEvent();
	thrust::host_vector<float> row_sums_host = row_sums;
	for(size_t i = 0; i < R;++i){
		colors[i] = row_sums_host[i];
	}
	//printf("repacked color[0]: %f\n",colors[0]);
	//ev3.AfterEvent();
	//ev3.Display();
}


void color_pack_test(){
	unsigned int R = 10777;     // number of rows
	unsigned int C = 2062;     // number of columns
	unsigned int RC = R*C;
	float *colors = new float[R];
	std::vector<int> indices;
	{
		indices.push_back(1);
		indices.push_back(5);
		indices.push_back(7);
	}
	float *data = new float[RC];
	{
		for(int i = 0; i < RC; ++i){
			data[i] = i%C;
		}
	}

	color_pack(indices,R,C,colors,data);
	printf("c: %f\n",colors[0]);
	printf("c: %f\n",colors[R-1]);
	delete [] data;
}

int thrust_main(void)
{
  int R = 10777;     // number of rows
  int C = 2062;     // number of columns
  int RC = R*C;

    // initialize data
  thrust::device_vector<float> array(R * C);
  thrust::host_vector<float> array_host(R * C);
  printf("created array: %d\n",array.size());

  for (size_t i = 0; i < array.size(); i++){
	  array_host[i] = i%C;
  }
  
  printf("host array filled\n");
  array = array_host;

  printf("a\n");
  // allocate storage for row sums and indices
  thrust::device_vector<float> row_sums(R);
  thrust::device_vector<float> row_indices(R);
  
  printf("b\n");
  // compute row sums by summing values with equal row indices
  thrust::reduce_by_key
    (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(C)),
     thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(C)) + (R*C),
     array.begin(),
     row_indices.begin(),
     row_sums.begin(),
     thrust::equal_to<float>(),
     thrust::plus<float>());
  
  thrust::host_vector<float> row_sums_host = row_sums;
  printf("c: %f\n",row_sums_host[0]);
  printf("c: %f\n",row_sums_host[R-1]);
  // print data 
  /*for(int i = 0; i < R; i++)
  {
    std::cout << "[ ";
    for(int j = 0; j < C; j++)
      std::cout << array[i * C + j] << " ";
    std::cout << "] = " << row_sums[i] << "\n";
  }*/

  return 0;
}
void cuda_main(){
	//memory_test_copy_pinned();
	//thrust_main();
	color_pack_test();
}
